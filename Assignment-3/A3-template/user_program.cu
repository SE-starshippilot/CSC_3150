﻿#include "virtual_memory.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ void user_program(VirtualMemory *vm, uchar *input, uchar *results,
                             int input_size) {

  for (int i = 0; i < input_size; i++){
    vm_write(vm, i, input[i]);
    printf("Writing value %d to address %d", input[i], i);
  }

  for (int i = input_size - 1; i >= input_size - 32769; i--){
    int value = vm_read(vm, i);
    printf("Read value %d from %d", value, i);
  }

  vm_snapshot(vm, results, 0, input_size);
}
