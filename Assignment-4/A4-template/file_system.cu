#include "hip/hip_runtime.h"
﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define CREATE_TIME_ATTR_OFFSET 2
#define MODIFY_TIME_ATTR_OFFSET 4
#define STARTBLK_ATTR_OFFSET 6
#define SIZE_ATTR_OFFSET 8
#define NAME_ATTR_OFFSET 12
#define CREATE_TIME_ATTR_LENGTH 2
#define MODIFY_TIME_ATTR_LENGTH 2
#define STARTBLK_ATTR_LENGTH 2
#define SIZE_ATTR_LENGTH 4
#define FCB_VALID 0b10000000
#define FCB_INVALID 0b00000000

__device__ __managed__ u32 gtime = 0; // increasing. larger means newer
__device__ __managed__ int gfilenum = 0;

__device__ void fcb_init(FileSystem* fs) {
  for (u32 i = 0; i < fs->FCB_ENTRIES; i++) {
    set_file_attr(fs, i, 0, 1, FCB_INVALID);// MSB in the first byte of FCB is valid bit. 0 indicates invalid.
  }
}

__device__ void superblock_init(FileSystem* fs) {
  // Initialize superblock. In my implementation, 0 means free and 1 means used.
  for (uchar i = 0; i < fs->SUPERBLOCK_SIZE; i++) {
    fs->volume[i] = 0;
  }
}

__device__ void fs_init(FileSystem* fs, uchar* volume, int SUPERBLOCK_SIZE,
  int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
  int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE,
  int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

  // init superblock
  superblock_init(fs);

  // init file control block
  fcb_init(fs);
}

__device__ int is_same_str(char* str1, char* str2) {
  /* Compare two strings. Return 1 if they are the same. */
  while (*str1 != '\0' && *str2 != '\0') {
    if (*str1 != *str2) return 0;
    str1++;
    str2++;
  }
  if (*str1 == '\0' && *str2 == '\0') return 1;
  else return 0;
}

__device__ char* get_file_attr(FileSystem* fs, u32 fp, int attr_offset) {
  u32 fcb_attr_addr = fs->SUPERBLOCK_SIZE + fp * fs->FCB_SIZE + attr_offset;
  int file_name_len = 0;
  while (fs->volume[fcb_attr_addr + file_name_len] != '\0') {
    file_name_len++;
  }
  file_name_len++;
  char* file_name = (char*)malloc(file_name_len);
  memcpy(file_name, fs->volume + fcb_attr_addr, file_name_len);
  return file_name;
}

__device__ int get_file_attr(FileSystem* fs, u32 fp, int attr_offset, int attr_length) {
  /* Read file attribute from FCB. */
  u32 fcb_attr_addr = fs->SUPERBLOCK_SIZE + fp * fs->FCB_SIZE + attr_offset;
  printf("[Read Attr from addr %d, length %d]\n", fcb_attr_addr, attr_length);
  int result = 0;
  for (int i = 0; i < attr_length; i++) {
    printf("reading byte %d:\t curr_result:%d\t", i, result);
    result = result << 8;
    printf("result << 8: %d\t curr_byte: %d\t", result, (int)fs->volume[fcb_attr_addr + i]);
    result += (int)fs->volume[fcb_attr_addr + i];
    printf("result after shifting: %d\n", result);
  }
  return result;
}

__device__ void set_file_attr(FileSystem* fs, u32 fp, int attr_offset, int attr_length, int value) {
  /* Set file attribute. */
  u32 fcb_attr_addr = fs->SUPERBLOCK_SIZE + fp * fs->FCB_SIZE + attr_offset;
  for (int i = attr_length - 1; i >= 0; i--) {
    fs->volume[fcb_attr_addr + i] = value & 0xFF;
    value = value >> 8;
  }
}

__device__ void set_file_attr(FileSystem* fs, u32 fp, int attr_offset, char* value) {
  /* Set file attribute. This reloaded function is for setting file name only. */
  int count = 0;
  int fcb_attr_addr = fs->SUPERBLOCK_SIZE + fp * fs->FCB_SIZE + attr_offset;
  while (value != '\0') {
    fs->volume[fcb_attr_addr + count] = value[count];
    count++;
    if (count == fs->MAX_FILENAME_SIZE) break;
  }
}

__device__ FCBQuery search_file(FileSystem* fs, char* s) {
  /* Find the FCB of the file with name s.
   * Return a query result containing the fcb index (if found) and first empty fcb index (if found).
   */
  FCBQuery ret_val = { -1, -1 };
  for (u32 i = 0; i < fs->FCB_ENTRIES; i++) {
    if (get_file_attr(fs, i, 0, 1) == FCB_VALID) { // valid bit is set
      if (is_same_str(s, (char*)fs->volume + fs->SUPERBLOCK_SIZE + i * fs->FCB_SIZE + 1)) {
        ret_val.FCB_index = i;
      }
    }
    else if (ret_val.empty_index == -1) {
      ret_val.empty_index = i;
    }
    if (ret_val.FCB_index != -1 && ret_val.empty_index != -1) break;
  }
  return ret_val;
}

__device__ u32 get_file_base_addr(FileSystem* fs, u32 fp) {
  /* Given a file pointer, return the base address of the file*/
  u32 file_start_block = get_file_attr(fs, fp, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH);
  return fs->volume[fs->FILE_BASE_ADDRESS + file_start_block * fs->STORAGE_BLOCK_SIZE];
}

__device__ u32 get_block_idx(FileSystem* fs, u32 addr) {
  /* Given an address(in the volume), return the corresponding block ID*/
  printf("addr=%d; base_addr=%d\n; delta=%d", addr, fs->FILE_BASE_ADDRESS, addr - fs->FILE_BASE_ADDRESS);
  return (addr - fs->FILE_BASE_ADDRESS) / fs->STORAGE_BLOCK_SIZE;
}

__device__ void vcb_set(FileSystem* fs, int fp, int val) {
  /* Set the corresponding VCB bits to 0 */
  int file_start_block = get_file_attr(fs, fp, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH);
  int file_size = get_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
  u32 file_start_block_idx = get_file_base_addr(fs, fp);
  int file_end_block = get_block_idx(fs, file_start_block_idx + file_size - 1);
  int block_start_byte = file_start_block / 8, block_end_byte = file_end_block / 8, block_start_offset = file_start_block % 8, block_end_offset = file_end_block % 8;
  if (val == 0) {
    fs->volume[block_start_byte] &= (0xff >> (8 - block_start_offset)) << (8 - block_start_offset);
    fs->volume[block_end_byte] &= 0xff >> (block_end_offset + 1);
  }
  else {
    fs->volume[block_start_byte] |= 0xff >> block_start_offset;
    fs->volume[block_end_byte] |= (0xff >> (7 - block_end_offset)) << (7 - block_end_offset);
  }
  for (int i = block_start_byte + 1; i < block_end_byte; i++) fs->volume[i] = val;
}

__device__ int count_vacant_bits(int VCB_Byte) {
  int count = 0;
  while (VCB_Byte) {
    count++;
    VCB_Byte &= VCB_Byte - 1;
  }
  return count;
}

__device__ int has_enough_space(FileSystem* fs, int block_size) {
  /* Check if there is enough space to put $block_size blocks*/
  int used_blocks = 0;
  for (int i = 0; i < fs->SUPERBLOCK_SIZE; i++)
    used_blocks += count_vacant_bits(fs->volume[i]);
  return fs->SUPERBLOCK_SIZE * 8 - used_blocks >= block_size;
}

__device__ int move_file(FileSystem* fs, u32 fp, int new_start_block_idx) {
  /* move file and return the next vacant block id after moving the file*/
  u32 old_file_base_addr = get_file_base_addr(fs, fp);
  u32 new_file_base_addr = fs->FILE_BASE_ADDRESS + new_start_block_idx * fs->STORAGE_BLOCK_SIZE;
  vcb_set(fs, fp, 0); // first, clear the original VCB bits
  set_file_attr(fs, fp, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH, new_start_block_idx);
  vcb_set(fs, fp, 1); // then, set the new VCB bits
  int file_size = get_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
  for (int i = 0; i < file_size; i++) {
    fs->volume[new_file_base_addr + i] = fs->volume[old_file_base_addr + i];
  }
  int file_block_size = ceil((float)file_size / fs->STORAGE_BLOCK_SIZE);
  return new_start_block_idx + file_block_size;
}

__device__ int fs_compress(FileSystem* fs) {
  /* Compress volume and retrun the first vacant block's index*/
  int next_vacant_block_idx = 0, prev_lowset_start_block_idx = 0;
  for (int i = 0; i < gfilenum; i++) {
    int curr_lowset_start_block_idx = 8 * fs->SUPERBLOCK_SIZE;
    int curr_lowest_start_block_fp;
    for (int j = 0; j < fs->FCB_ENTRIES; j++) {
      if (get_file_attr(fs, j, 0, 1) == FCB_INVALID) continue;
      int file_start_block_idx = get_file_attr(fs, j, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH);
      if (file_start_block_idx < prev_lowset_start_block_idx) continue;
      if (file_start_block_idx < curr_lowset_start_block_idx) { 
        curr_lowset_start_block_idx = file_start_block_idx; 
        curr_lowest_start_block_fp = j;  
      }
    }
    if (curr_lowset_start_block_idx != next_vacant_block_idx) {
      prev_lowset_start_block_idx = curr_lowset_start_block_idx;
      next_vacant_block_idx = move_file(fs, curr_lowest_start_block_fp, next_vacant_block_idx);
    }
  }
  return next_vacant_block_idx;
}

__device__ u32 fs_allocate(FileSystem* fs, int block_num) {
  /* Return the first place that can hold $block_num blocks*/
  /* Use first fit algirthm. First, check if the volume has enough space.*/
  if (!has_enough_space(fs, block_num)) return -1;
  /* If there are enough space */
  int count = 0;
  int t_block_idx = 0;
  /* Use first fit to find the starting block index*/
  for (int i = 0; i < fs->SUPERBLOCK_SIZE; i++) {
    unsigned int curr_byte = (unsigned int)fs->volume[i];
    for (int j = 7; j >= 0; j--) {
      if (curr_byte & (1 << j)) count = 0;
      else {
        if (count == 0) t_block_idx = i * 8 + 7 - j;
        count++;
        if (count == block_num) return t_block_idx;
      }
    }
  }
  /* If no such block is found, compress volume*/
  return fs_compress(fs);
}

__device__ u32 fs_open(FileSystem* fs, char* s, int op)
{
  /* Implement open operation here */
  FCBQuery query = search_file(fs, s);
  if (query.FCB_index != -1) return query.FCB_index;
  if (op == G_READ) {
    return -1;
  }
  else if (op == G_WRITE) {
    if (query.empty_index == -1) return -1; // maximum # of files reached
    else {
      set_file_attr(fs, query.empty_index, NAME_ATTR_OFFSET, s); // set file name
      set_file_attr(fs, query.empty_index, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH, 0); // set file size
      set_file_attr(fs, query.empty_index, CREATE_TIME_ATTR_OFFSET, CREATE_TIME_ATTR_LENGTH, gtime); // set create time
      set_file_attr(fs, query.empty_index, MODIFY_TIME_ATTR_OFFSET, MODIFY_TIME_ATTR_LENGTH, gtime); // set modify time
      gtime++;
      gfilenum++;
      printf("Currently there are %d files", gfilenum);
      return query.empty_index;
    }
  }
  else {
    printf("Invalid operation code.\n");
  }

}

__device__ void fs_read(FileSystem* fs, uchar* output, u32 size, u32 fp)
{
  /* Implement read operation here */
  if (fp == -1) {
    printf("File not found.\n");
    return;
  }
  int file_size = get_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
  if (size > file_size) {
    printf("Read size exceeds file size.\n");
    return;
  }
  u32 file_start_block_idx = get_file_base_addr(fs, fp);
  for (int i = 0; i < size; i++)
    output[i] = fs->volume[file_start_block_idx + i];
}

__device__ u32 fs_write(FileSystem* fs, uchar* input, u32 size, u32 fp)
{
  /* Implement write operation here */
  if (fp == -1) {
    printf("Invalid fp.\n");
    return -1;
  }
  u32 orgn_file_size = get_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
  int orgn_pos_max_size = floor((float)orgn_file_size / fs->STORAGE_BLOCK_SIZE) * fs->STORAGE_BLOCK_SIZE; // the maximum size the previous location can hold 
  u32 new_file_base_addr = get_file_base_addr(fs, fp); // set the new file base address to the original one
  u32 new_file_start_block = get_file_attr(fs, fp, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH); // as well as the new file start block
  printf("originally file is %d Bytes.\n", orgn_file_size);
  printf("The original space can store up to %d Bytes of file.\n", orgn_pos_max_size);
  if (size < orgn_file_size) { // If the new size is smaller than the original file, clear VCB and set according to new size
    vcb_set(fs, fp, 0); // clear the VCB bits
    set_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH, size); // update file size
    vcb_set(fs, fp, 1); // set the VCB bits
  }
  else if (size > orgn_pos_max_size)
  { // need to reallocate space for file.Clear previous VCB and allocate new space.
    int new_block_num = ceil((float)size / fs->STORAGE_BLOCK_SIZE);
    vcb_set(fs, fp, 0);
    set_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH, size); // update file size
    new_file_start_block = fs_allocate(fs, new_block_num);
    set_file_attr(fs, fp, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH, new_file_start_block); // update file start block
    vcb_set(fs, fp, 1);
  }
  else {
    set_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH, size); // update file size
  }
  // write $size bytes to the new starting position 
  // set_file_attr(fs, fcb_base_addr + SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH, size); // set file size
  for (int i = 0; i < size; i++) // write file content
    fs->volume[new_file_base_addr + i] = input[i];
  set_file_attr(fs, fp, MODIFY_TIME_ATTR_OFFSET, MODIFY_TIME_ATTR_LENGTH, gtime); // set modify time
  gtime++;
}

__device__ void fs_gsys(FileSystem* fs, int op)
{
  /* Implement LS_D and LS_S operation here */
  if (op == LS_D) {
    /* Since at each time only one file is modified, two files cannot have the same modification time.*/
    printf("===sort by modified time===\n");
    int prev_youngest_modtime = gtime;
    for (int i = 0; i < gfilenum; i++) {
      int curr_youngest_modtime = -1;
      char* curr_file_name;
      for (int j = 0; j < fs->FCB_ENTRIES; j++) {
        if (get_file_attr(fs, j, 0, 1) == FCB_INVALID) continue;
        int file_modtime = get_file_attr(fs, j, MODIFY_TIME_ATTR_OFFSET, MODIFY_TIME_ATTR_LENGTH);
        if (file_modtime > prev_youngest_modtime) continue;
        if (file_modtime > curr_youngest_modtime) {
          curr_youngest_modtime = file_modtime;
          curr_file_name = get_file_attr(fs, j, NAME_ATTR_OFFSET);
        }
      }
      printf("%-20s", curr_file_name);
      prev_youngest_modtime = curr_youngest_modtime;
    }
  }
  else if (op == LS_S) {
    int prev_max_size = fs->MAX_FILE_SIZE, prev_oldest_create_time = -1;
    for (int i = 0; i < gfilenum; i++) {
      int curr_max_size = -1, curr_oldest_create_time = gtime;
      char* curr_file_name;
      for (int j = 0; j < fs->FCB_ENTRIES; j++) {
        if (get_file_attr(fs, j, 0, 1) == FCB_INVALID) continue;
        int file_size = get_file_attr(fs, j, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
        int file_create_time = get_file_attr(fs, j, CREATE_TIME_ATTR_OFFSET, CREATE_TIME_ATTR_LENGTH);
        if (file_size > prev_max_size || (file_size == prev_max_size) && file_create_time <= prev_oldest_create_time) continue;
        if (file_size > curr_max_size || (file_size == curr_max_size && file_create_time < curr_oldest_create_time)) {
          curr_max_size = file_size;
          curr_oldest_create_time = file_create_time;
          curr_file_name = get_file_attr(fs, j, NAME_ATTR_OFFSET);
        }
      }
      printf("%-20s\t %d", curr_file_name, curr_max_size);
      prev_max_size = curr_max_size;
      prev_oldest_create_time = curr_oldest_create_time;
    }
  }
  else {
    printf("Invalid operation code [%d]\n", op);
  }
}

__device__ void fs_gsys(FileSystem* fs, int op, char* s)
{
  /* Implement rm operation here */
  if (op != RM) {
    printf("Invalid operation [%d].\n", op);
    return;
  }
  FCBQuery query = search_file(fs, s);
  if (query.FCB_index == -1) {
    printf("No file named %s to delete.\n", s);
  }
  vcb_set(fs, query.FCB_index, 0);
  set_file_attr(fs, query.FCB_index, 0, 1, FCB_INVALID);
  gfilenum--;
}
