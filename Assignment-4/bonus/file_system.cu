#include "hip/hip_runtime.h"
﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>

#define CREATE_TIME_ATTR_OFFSET 2
#define MODIFY_TIME_ATTR_OFFSET 4
#define STARTBLK_ATTR_OFFSET 6
#define SIZE_ATTR_OFFSET 8
#define NAME_ATTR_OFFSET 12
#define CREATE_TIME_ATTR_LENGTH 2
#define MODIFY_TIME_ATTR_LENGTH 2
#define STARTBLK_ATTR_LENGTH 2
#define PARDIR_ATTR_LENGTH 2
#define SIZE_ATTR_LENGTH 4
#define FCB_VALID 0b10000000
#define FCB_INVALID 0b00000000
#define DIR 0b11000000
#define FP_INVALID 1024
#define PARENT_DIR(x) x&0x03ff
#define DIR_LEVEL(x) (x&0x30) >> 4 

__device__ __managed__ u32 gtime = 0;       // increasing. larger means newer
__device__ __managed__ u32 gfilenum = 0;    // number of files present in the file system
__device__ __managed__ u32 glastblock = 0;  // used in next-fit algorithm
__device__ __managed__ u32 gcwd = 0;        // current working directory. Default is root. root directory is always at fcb#0

__device__ void fcb_init(FileSystem* fs) {
  for (u32 i = 0; i < fs->FCB_ENTRIES; i++) {
    set_file_attr(fs, i, 0, 1, FCB_INVALID);// MSB in the first byte of FCB is valid bit. 0 indicates invalid.
  }
}

__device__ void superblock_init(FileSystem* fs) {
  // Initialize superblock. In my implementation, 0 means free and 1 means used.
  for (int i = 0; i < fs->SUPERBLOCK_SIZE; i++) {
    fs->volume[i] = (uchar)0x00;
  }
}

__device__ void fs_init(FileSystem* fs, uchar* volume, int SUPERBLOCK_SIZE,
  int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
  int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE,
  int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;

  // init superblock
  superblock_init(fs);

  // init file control block
  fcb_init(fs);

  // create the fcb for root directory
}

__device__ int str_cmp(char* str1, char* str2) {
  /* Compare two strings. Return 1 if they are the same. */
  while (*str1 != '\0' && *str2 != '\0') {
    if (*str1 != *str2) return 0;
    str1++;
    str2++;
  }
  if (*str1 == '\0' && *str2 == '\0') return 1;
  else return 0;
}

__device__ int str_len(const char* str) {
  /* Return the length of a string. */
  const char* s;
  for (s = str; *s; ++s);
  return(s - str);
}

__device__ int str_cpy(char* str1, const char* str2) {
  int len_1 = str_len(str1);
  int len_2 = str_len(str2);
}

__device__ void str_cat(char* str1, char* str2) {

}

__device__ char* get_file_attr(FileSystem* fs, u32 fp, int attr_offset) {
  u32 fcb_attr_addr = fs->SUPERBLOCK_SIZE + fp * fs->FCB_SIZE + attr_offset;
  char* file_name = (char*)fs->volume + fcb_attr_addr;
  return file_name;
}

__device__ int get_file_attr(FileSystem* fs, u32 fp, int attr_offset, int attr_length) {
  /* Read file attribute from FCB. */
  int result = 0;
  memcpy(&result, fs->volume + fs->SUPERBLOCK_SIZE + fp * fs->FCB_SIZE + attr_offset, attr_length);
  return result;
}

__device__ void set_file_attr(FileSystem* fs, u32 fp, int attr_offset, int attr_length, int value) {
  /* Set file attribute. */
  memcpy(fs->volume + fs->SUPERBLOCK_SIZE + fp * fs->FCB_SIZE + attr_offset, &value, attr_length);
}

__device__ void set_file_attr(FileSystem* fs, u32 fp, int attr_offset, int attr_length, char* value) {
  /* Set file attribute. This reloaded function is for setting file name only. */
  memcpy(fs->volume + fs->SUPERBLOCK_SIZE + fp * fs->FCB_SIZE + attr_offset, value, attr_length);
  memset(fs->volume + fs->SUPERBLOCK_SIZE + fp * fs->FCB_SIZE + attr_offset + attr_length, 0, 1);
}

__device__ FCBQuery search_file(FileSystem* fs, char* s) {
  /* Find the FCB of the file with name s.
   * Return a query result containing the fcb index (if found) and first empty fcb index (if found).
   */
  FCBQuery ret_val = { FP_INVALID, FP_INVALID };
  int valid_fcb_traversed = 0;
  for (u32 i = 0; i < fs->FCB_ENTRIES; i++) {
    if (get_file_attr(fs, i, 0, 1) == FCB_VALID) { // valid bit is set
      valid_fcb_traversed++;
      if (str_cmp(s, get_file_attr(fs, i, NAME_ATTR_OFFSET))) {
        ret_val.FCB_index = i;
        break;
      }
    }
    else if (ret_val.empty_index == FP_INVALID) {
      ret_val.empty_index = i;
    }
    if (valid_fcb_traversed == gfilenum && ret_val.empty_index != FP_INVALID) break;
  }
  return ret_val;
}

__device__ u32 get_file_base_addr(FileSystem* fs, u32 fp) {
  /* Given a file pointer, return the base address of the file*/
  u32 file_start_block = get_file_attr(fs, fp, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH);
  return fs->FILE_BASE_ADDRESS + file_start_block * fs->STORAGE_BLOCK_SIZE;
}

__device__ u32 get_block_idx(FileSystem* fs, u32 addr) {
  /* Given an address(in the volume), return the corresponding block ID*/
  return (addr - fs->FILE_BASE_ADDRESS) / fs->STORAGE_BLOCK_SIZE;
}

__device__ u32 get_file_end_block(FileSystem* fs, u32 fp) {
  /* Given a file pointer, return the end block of the file*/
  u32 file_start_block = get_file_attr(fs, fp, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH);
  u32 file_size = get_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
  u32 file_block_count = ceil((float)file_size / fs->STORAGE_BLOCK_SIZE);
  return file_start_block + file_block_count - 1;
}

__device__ void vcb_set(FileSystem* fs, int fp, int val) {
  /* Set the corresponding VCB bits to 0 */
  int file_size = get_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
  if (file_size == 0) return;
  int file_start_block = get_file_attr(fs, fp, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH);
  int file_end_block = get_file_end_block(fs, fp);
  int start_byte = file_start_block / 8, start_offset = file_start_block % 8, end_byte = file_end_block / 8, end_offset = file_end_block % 8;
  int start_mask = 0xff >> start_offset, end_mask = (0xff >> (7 - end_offset)) << (7 - end_offset);
  if (start_byte == end_byte) {
    int mask = start_mask & end_mask;
    if (val) {
      fs->volume[start_byte] |= mask;
    }
    else {
      fs->volume[start_byte] &= ~mask;
    }
  }
  else {
    if (val) {
      fs->volume[start_byte] |= start_mask;
      fs->volume[end_byte] |= end_mask;
    }
    else {
      fs->volume[start_byte] &= ~(start_mask);
      fs->volume[end_byte] &= ~(end_mask);
    }
    for (int i = start_byte + 1; i < end_byte; i++) fs->volume[i] = val;
  }
}

__device__ int move_file(FileSystem* fs, u32 fp, int new_start_block_idx) {
  /* move file and return the next vacant block id after moving the file*/
  u32 old_file_base_addr = get_file_base_addr(fs, fp);
  u32 new_file_base_addr = fs->FILE_BASE_ADDRESS + new_start_block_idx * fs->STORAGE_BLOCK_SIZE;
  vcb_set(fs, fp, 0); // first, clear the original VCB bits
  set_file_attr(fs, fp, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH, new_start_block_idx);
  vcb_set(fs, fp, 1); // then, set the new VCB bits
  int file_size = get_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
  memcpy(fs->volume + new_file_base_addr, fs->volume + old_file_base_addr, file_size);
  int file_end_block = get_file_end_block(fs, fp);
  return file_end_block + 1;
}

__device__ int fs_compress(FileSystem* fs) {
  /* Compress volume and retrun the first vacant block's index*/
  u32* fcb_arr = new u32[gfilenum];
  u32* startblk_arr = new u32[gfilenum];
  int files_found = 0;
  for (int i = 0; i < fs->FCB_ENTRIES; i++) {
    if (get_file_attr(fs, i, 0, 1) == FCB_VALID) {
      fcb_arr[files_found] = i;
      startblk_arr[files_found] = get_file_attr(fs, i, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH);
      files_found++;
    }
    if (files_found == gfilenum) break;
  }
  for (int i = 0; i < gfilenum; i++) {
    int curr_min = i;
    for (int j = i + 1; j < gfilenum; j++) {
      if (startblk_arr[curr_min] > startblk_arr[j]) {
        curr_min = j;
      }
    }
    int tmp = startblk_arr[i];
    startblk_arr[i] = startblk_arr[curr_min];
    startblk_arr[curr_min] = tmp;
    tmp = fcb_arr[i];
    fcb_arr[i] = fcb_arr[curr_min];
    fcb_arr[curr_min] = tmp;
  }
  int prev_smallest_start_block = 0;
  for (int i = 0; i < gfilenum; i++) {
    if (startblk_arr[i] != prev_smallest_start_block) {
      prev_smallest_start_block = move_file(fs, fcb_arr[i], prev_smallest_start_block);
    }
  }
  delete[] fcb_arr;
  delete[] startblk_arr;
  return prev_smallest_start_block;
}

__device__ u32 fs_allocate(FileSystem* fs, int block_num) {
  /* Return the index of first block that can hold $block_num blocks*/
  /* Use first fit algirthm. First, check if the volume has enough space.*/
  /* If there are enough space */
  int count = 0;
  int t_block_idx = 0;
  /* Use first fit to find the starting block index*/
  for (int i = glastblock; i < fs->SUPERBLOCK_SIZE * 8; i++) {
    int curr_block_byte = i / 8, curr_block_offset = i % 8;
    int curr_block_status = fs->volume[curr_block_byte] & (1 << (7 - curr_block_offset));
    if (curr_block_status == 0) {
      count++;
      if (count == 1) t_block_idx = i;
      if (count == block_num) {
        glastblock = i;
        return t_block_idx;
      }
    }
    else {
      count = 0;
    }
  }
  /* If no such block is found, compress volume*/
  glastblock = fs_compress(fs);
  if (glastblock + block_num > fs->SUPERBLOCK_SIZE * 8) {
    printf("No enough space to allocate %d blocks\n", block_num);
    return fs->SUPERBLOCK_SIZE * 8;
  }
  return glastblock;
}

__device__ u32 fs_open(FileSystem* fs, char* s, int op)
{
  /* Implement open operation here */
  int file_name_length = str_len(s);
  if (file_name_length > fs->MAX_FILENAME_SIZE) return FP_INVALID << 1;
  FCBQuery query = search_file(fs, s);
  int ret_val = query.FCB_index;
  if (op == G_WRITE) {
    if (ret_val == FP_INVALID) {
      if (query.empty_index == FP_INVALID) {
        printf("Maximum #file reached.\n");
      }
      else {
        ret_val = query.empty_index;
        set_file_attr(fs, query.empty_index, 0, 1, FCB_VALID);
        set_file_attr(fs, query.empty_index, NAME_ATTR_OFFSET, file_name_length, s); // set file name
        set_file_attr(fs, query.empty_index, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH, 0); // set file size
        set_file_attr(fs, query.empty_index, CREATE_TIME_ATTR_OFFSET, CREATE_TIME_ATTR_LENGTH, gtime); // set create time
        set_file_attr(fs, query.empty_index, MODIFY_TIME_ATTR_OFFSET, MODIFY_TIME_ATTR_LENGTH, gtime); // set modify time
        gtime++;
        gfilenum++;
      }
    }
  }
  else if (op != G_READ) {
    printf("Invalid operation code.\n");
    ret_val = FP_INVALID;
  }
  return (ret_val << 1) + op;
}

__device__ void fs_read(FileSystem* fs, uchar* output, u32 size, u32 fp)
{
  /* Implement read operation here */
  int mode = fp & 1;
  fp >>= 1;
  if (fp == fs->FCB_ENTRIES || mode != G_READ) {
    printf("File not found.\n");
    return;
  }
  int file_size = get_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
  if (size > file_size) {
    printf("Read size exceeds file size.\n");
    return;
  }
  u32 file_base_addr = get_file_base_addr(fs, fp);
  memcpy(output, fs->volume + file_base_addr, size);
}

__device__ u32 fs_write(FileSystem* fs, uchar* input, u32 size, u32 fp)
{
  /* Implement write operation here */
  /* return 1 means error, 0 means success*/
  int mode = fp & 1;
  fp >>= 1;
  if (fp == fs->FCB_ENTRIES || mode != G_WRITE) {
    printf("Invalid fp.\n");
    return 1;
  }
  u32 orgn_file_size = get_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
  int orgn_pos_max_size = floor((float)orgn_file_size / fs->STORAGE_BLOCK_SIZE) * fs->STORAGE_BLOCK_SIZE; // the maximum size the previous location can hold 
  u32 new_file_start_block;
  if (size < orgn_file_size) { // If the new size is smaller than the original file, clear VCB and set according to new size
    vcb_set(fs, fp, 0); // clear the VCB bits
    set_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH, size); // update file size
    vcb_set(fs, fp, 1); // set the VCB bits
  }
  else if (size > orgn_pos_max_size)
  { // need to reallocate space for file.Clear previous VCB and allocate new space.
    vcb_set(fs, fp, 0);
    set_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH, size); // update file size
    int new_block_num = ceil((float)size / fs->STORAGE_BLOCK_SIZE);
    new_file_start_block = fs_allocate(fs, new_block_num);
    if (new_file_start_block == fs->SUPERBLOCK_SIZE * 8) {
      printf("No enough space.\n");
      // roll back
      set_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH, orgn_file_size);
      vcb_set(fs, fp, 1);
      return 1;
    }
    set_file_attr(fs, fp, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH, new_file_start_block); // update file start block
    vcb_set(fs, fp, 1);
  }
  else {
    set_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH, size); // update file size
  }
  u32 new_file_base_addr = get_file_base_addr(fs, fp);
  // write $size bytes to the new starting position 
  memcpy(fs->volume + new_file_base_addr, input, size);
  set_file_attr(fs, fp, MODIFY_TIME_ATTR_OFFSET, MODIFY_TIME_ATTR_LENGTH, gtime); // set modify time
  gtime++;
  return 0;
}

__device__ void fs_gsys(FileSystem* fs, int op)
{
  /* Implement LS_D and LS_S operation here */
  switch (op)
  {
  case LS_D:
    int* fcb_arr = new int[gfilenum];
    int* modtime_arr = new int[gfilenum];
    int files_found = 0;
    for (int i = 0; i < fs->FCB_ENTRIES; i++) {
      if (get_file_attr(fs, i, 0, 1) == FCB_VALID) {
        fcb_arr[files_found] = i;
        modtime_arr[files_found] = get_file_attr(fs, i, MODIFY_TIME_ATTR_OFFSET, MODIFY_TIME_ATTR_LENGTH);
        files_found++;
      }
      if (files_found == gfilenum) break;
    }
    for (int i = 0; i < gfilenum; i++) {
      if (gfilenum == 0) break;
      int curr_max = i;
      for (int j = i + 1; j < gfilenum; j++) {
        if (modtime_arr[curr_max] < modtime_arr[j]) {
          curr_max = j;
        }
      }
      int tmp = modtime_arr[i];
      modtime_arr[i] = modtime_arr[curr_max];
      modtime_arr[curr_max] = tmp;
      tmp = fcb_arr[i];
      fcb_arr[i] = fcb_arr[curr_max];
      fcb_arr[curr_max] = tmp;
    }
    printf("===sort by modified time===\n");
    for (int i = 0; i < gfilenum; i++) {
      char is_dir = (get_file_attr(0, fcb_arr[i], 0, 1) == DIR) ? ' ' : 'd';
      printf("%-20s\t%c\n", get_file_attr(fs, fcb_arr[i], NAME_ATTR_OFFSET), modtime_arr[i], is_dir);
    }
    delete[] fcb_arr;
    delete[] modtime_arr;
    break;
  case LS_S:
    int* fcb_arr = new int[gfilenum];
    int* size_arr = new int[gfilenum];
    int files_found = 0;
    for (int i = 0; i < fs->FCB_ENTRIES; i++) {
      if (get_file_attr(fs, i, 0, 1) == FCB_VALID) {
        fcb_arr[files_found] = i;
        size_arr[files_found] = get_file_attr(fs, i, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
        files_found++;
      }
      if (files_found == gfilenum) break;
    }
    for (int i = 0; i < gfilenum; i++) {
      if (gfilenum == 1) break;
      int curr_max = i;
      for (int j = i + 1; j < gfilenum; j++) {
        if (size_arr[curr_max] < size_arr[j] ||
          (size_arr[curr_max] == size_arr[j] && (get_file_attr(fs, fcb_arr[curr_max], CREATE_TIME_ATTR_OFFSET, CREATE_TIME_ATTR_LENGTH) > get_file_attr(fs, fcb_arr[j], CREATE_TIME_ATTR_OFFSET, CREATE_TIME_ATTR_LENGTH)))
          ) {
          curr_max = j;
        }
      }
      int temp = size_arr[i];
      size_arr[i] = size_arr[curr_max];
      size_arr[curr_max] = temp;
      temp = fcb_arr[i];
      fcb_arr[i] = fcb_arr[curr_max];
      fcb_arr[curr_max] = temp;
    }
    printf("===sort by file size===\n");
    for (int i = 0; i < gfilenum; i++) {
      char is_dir = (get_file_attr(0, fcb_arr[i], 0, 1) == DIR) ? ' ' : 'd';
      printf("%-20s\t%-8d\t%c\n", get_file_attr(fs, fcb_arr[i], NAME_ATTR_OFFSET), get_file_attr(fs, fcb_arr[i], SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH), is_dir);
    }
    delete[] fcb_arr;
    delete[] size_arr;
    break;
  case CD_P:
    int raw_pd = get_file_attr(fs, gcwd, 0, 2);
    gcwd = PARENT_DIR(raw_pd);
    break;
  case PWD:
    int tcwd = gcwd;
    int tinfo = get_file_attr(fs, tcwd, 0, 1);
    int tlevel = DIR_LEVEL(tinfo);
    int* working_dir = new int[tlevel];
    for (int i = 0; i < tlevel; i++) {
      working_dir[i] = tcwd;
      int tparent = get_file_attr(fs, tcwd, 0, PARDIR_ATTR_LENGTH);
      tcwd = PARENT_DIR(tparent);
    }
    for (int i = tlevel - 1; i >= 0; i--)
      printf("/%s", get_file_attr(fs, working_dir[i], NAME_ATTR_OFFSET));
    printf("\n");
    delete[] working_dir;
    break;
  default:
    printf("Invalid operation code [%d]\n", op);
    break;
  }
}

__device__ void fs_gsys(FileSystem* fs, int op, char* s)
{
  /* Implement rm operation here */
  FCBQuery query = search_file(fs, s);
  if (query.FCB_index == FP_INVALID && op != MKDIR) {
    printf("No file named %s");
    return;
  }
  switch (op) {
  case RM:
    if (get_file_attr(fs, query.FCB_index, 0, 1) == DIR) {
      printf("Cannot delete a directory using RM,\n");
      return;
    }
    vcb_set(fs, query.FCB_index, 0);
    set_file_attr(fs, query.FCB_index, 0, 1, FCB_INVALID);
    gfilenum--;
    break;
  case RM_RF:
    if (get_file_attr(fs, query.FCB_index, 0, 1) != DIR) {
      printf("Cannot delete a file using RM_RF.\n");
      return;
    }
    else if (query.FCB_index == 0) {
      printf("Cannot remove root directory.\n");
      return;
    }
    int dir_size = get_file_attr(fs, query.FCB_index, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH), read_size = 0;
    uchar* dir_content = new uchar[dir_size];
    fs_read(fs, dir_content, dir_size, query.empty_index << 1);
    while (read_size != dir_size) {
      char* t_name = (char*)dir_content;
      FCBQuery t_query = search_file(fs, t_name);
      int t_fp = t_query.FCB_index;
      if (get_file_attr(fs, t_fp, 0, 1) == DIR) {
        fs_gsys(fs, RM_RF, t_name); // recursively remove all the files within the directory
      }
      else {
        fs_gsys(fs, RM, t_name);
      }
      read_size += str_len(t_name);
    }
    vcb_set(fs, query.FCB_index, 0); // clear vcb bits
    set_file_attr(fs, query.FCB_index, 0, 1, FCB_INVALID); // invalidate fcb entry
    gfilenum--; // decrease file numbers
    delete[] dir_content;
    break;
  case CD:
    if (get_file_attr(fs, query.FCB_index, 0, 1) != DIR) {
      printf("Cannot CD into a file.\n");
    }
    gcwd = query.FCB_index;
    break;
  default:
    printf("Invalid operation code [%d]\n", op);
  }
}

__device__ void fs_diagnose(FileSystem* fs, u32 fp) {
  char* file_name = get_file_attr(fs, fp, NAME_ATTR_OFFSET);
  short file_modtime = get_file_attr(fs, fp, MODIFY_TIME_ATTR_OFFSET, MODIFY_TIME_ATTR_LENGTH);
  int file_size = get_file_attr(fs, fp, SIZE_ATTR_OFFSET, SIZE_ATTR_LENGTH);
  short file_createtime = get_file_attr(fs, fp, CREATE_TIME_ATTR_OFFSET, CREATE_TIME_ATTR_LENGTH);
  short file_startblock = get_file_attr(fs, fp, STARTBLK_ATTR_OFFSET, STARTBLK_ATTR_LENGTH);
  int file_endblock = get_file_end_block(fs, fp);
  printf("FCB Index:%-4d\tFile name:%-20s\tSize:%-10d\tStarts on block:%-5d\tEnds on block:%-5d\tTime created:%-5d\tTime modified:%-5d\n", fp, file_name, file_size, file_startblock, file_endblock, file_createtime, file_modtime);
}